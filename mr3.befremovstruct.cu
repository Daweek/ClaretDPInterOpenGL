#include "hip/hip_runtime.h"
/*Archivo mr3.cu que contiene el codigo para CUDA.
    Renderizacion por OpenGL-CUDA interoperability
    Nucleo del codigo para calcular la fuerza entre particulas
    Creado por: Martinez Noriega Edgar Josafat
*/
#define GL_ON
#define KER
//#define DP
#define INTEROP
//#define TIME_MEMORY
/////////////
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
// ***** CUDA includes
#include <hip/hip_runtime.h>
#include <nvcuvid.h>
#include <cudaGL.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#define NMAX      8192
#define NTHRE       512
#define ATYPE        8
#define ATYPE2    (ATYPE * ATYPE)
#define ThreadsPB 512
//////For NaCl Optminized if_kernel
#define NTHREOPT      512
#define NDIVBIT      4
#define NDIV      (1<<NDIVBIT)
#define NTHREOPT2    (NTHREOPT/NDIV)

typedef struct {
  float r[3];
  int atype;
} VG_XVEC;

typedef struct {
  float pol;
  float sigm;
  float ipotro;
  float pc;
  float pd;
  float zz;
} VG_MATRIX;


/////////GLOBAL Variables/////////////////////////////////////////
int   	*d_atypemat;
VG_XVEC *d_x=NULL;
VG_XVEC	*vec=NULL;
float 	*d_force=NULL;
float   *d_side,*d_sideh;
float   *d_amass,*d_vl;
float 	*d_ekin1;
float 	*d_ekin,*d_xs,*d_mtemp,*d_mpres;
float		*d_poss,*d_colr;

int mem_flg=0;
int mem_flg2=0;
int mem_sp=5;
int mem_cpu=0;
int flg1=0,flg2=0,flg3=0;

extern GLuint g_possVBO, g_colorVBO;
extern hipDeviceProp_t g_devprop;
extern struct hipGraphicsResource* g_strucPossVBOCUDA;
extern struct hipGraphicsResource* g_strucColorVBOCUDA;

__constant__
VG_MATRIX c_matrix[4]={[0].pol=1.250000,[0].sigm=2.340000,[0].ipotro=3.154574,[0].pc=0.072868,[0].pd=0.034699,[0].zz=1.000000,
	[1].pol=1.000000,[1].sigm=2.755000,[1].ipotro=3.154574,[1].pc=0.485784,[1].pd=0.602893,[1].zz=-1.000000,
	[2].pol=1.000000,[2].sigm=2.755000,[2].ipotro=3.154574,[2].pc=0.485784,[2].pd=0.602893,[2].zz=-1.000000,
	[3].pol=0.750000,[3].sigm=3.170000,[3].ipotro=3.154574,[3].pc=5.031334,[3].pd=10.106042,[3].zz=1.000000};

__constant__
float d_color_table[5][4]={ {0.35	,0.19	,0.19	,1.0},
														{0.19	,0.275,0.19	,1.0},
														{1.0	,0.4	,1.0	,1.0},
														{0.0	,0.8	,1.0	,1.0},
														{1.0	,1.0	,1.0	,1.0} };

//////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////
////////FORCE CALCULATION WITH GPU/////////////////////////////////////
//////////////////////////////////////////////////////////////////////
__global__
void update_coor_kernel(int n3, float *vl,VG_XVEC *cd,float *xs,
                        float *fc,float *side){
#ifdef KER
	int tid  = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < n3){
		vl[tid]   =  (vl[tid]*(1-(*xs))+fc[tid])/(1+(*xs));
    cd[tid/3].r[tid % 3]   +=   vl[tid];
		if (cd[tid/3].r[tid % 3] < 0 || cd[tid/3].r[tid % 3] > side[tid % 3]) vl[tid] *= -1;
	}
#endif
}
//////////////////////////////////////////////////////////////////////////
__device__ __inline__
void inter_if(float xj[3], float xi[3], float fi[3], int t, float xmax,
		float xmax1) {
#ifdef KER

	int k;
	float dn2, r, inr, inr2, inr4, inr8, d3, dr[3];
	float pb = (float) (0.338e-19 / (14.39 * 1.60219e-19)), dphir;

	dn2 = 0.0f;
	for (k = 0; k < 3; k++) {
		dr[k] = xi[k] - xj[k];
		dr[k] -= rintf(dr[k] * xmax1) * xmax;
		dn2 += dr[k] * dr[k];
	}
	r = sqrtf(dn2);
#if 1
	inr = 1.0f / r;
#elif 0
	if(dn2 != 0.0f) inr = 1.0f / r;
	else inr = 0.0f;
#elif 0
	if(dn2 == 0.0f) inr = 0.0f;
	else inr = 1.0f / r;
#else
	inr = 1.0f / r;
	if(dn2 == 0.0f) inr = 0.0f;
#endif
	inr2 = inr * inr;
	inr4 = inr2 * inr2;
	inr8 = inr4 * inr4;
	d3 = pb * c_matrix[t].pol
			* expf((c_matrix[t].sigm - r) * c_matrix[t].ipotro);
	dphir =
			(d3 * c_matrix[t].ipotro * inr - 6.0f * c_matrix[t].pc * inr8
					- 8.0f * c_matrix[t].pd * inr8 * inr2
					+ inr2 * inr * c_matrix[t].zz);
#if 1
	if (dn2 == 0.0f)
		dphir = 0.0f;
#endif
	for (k = 0; k < 3; k++)
		fi[k] += dphir * dr[k];
#endif
}

__global__
void nacl_kernel_if2(VG_XVEC *x, int n, int nat, float xmax, float *fvec) {
#ifdef KER
	int tid = threadIdx.x;
	int jdiv = tid / NTHREOPT2;
	int i = blockIdx.x * NTHREOPT2 + (tid & (NTHREOPT2 - 1)); // Same + (tid %16)
	int j, k;
	float xmax1 = 1.0f / xmax;
	int atypei;
	float xi[3];
	__shared__ VG_XVEC s_xj[NTHREOPT];
	__shared__ float s_fi[NTHREOPT][3];

	for (k = 0; k < 3; k++)
		s_fi[tid][k] = 0.0f;
	for (k = 0; k < 3; k++)
		xi[k] = x[i].r[k];
	atypei = x[i].atype * nat;
	int na;
	na = n / NTHREOPT;
	na = na * NTHREOPT;
	for (j = 0; j < na; j += NTHREOPT) {
		__syncthreads();
		s_xj[tid] = x[j + tid];
		__syncthreads();
#pragma unroll 16
		for (int js = jdiv; js < NTHREOPT; js += NDIV)
			inter_if(s_xj[js].r, xi, s_fi[tid], atypei + s_xj[js].atype, xmax,
					xmax1);
	}
	for (j = na + jdiv; j < n; j += NDIV) {
		inter_if(x[j].r, xi, s_fi[tid], atypei + x[j].atype, xmax, xmax1);
	}
#if NTHREOPT>=512 && NTHREOPT2<=256
	__syncthreads();
	if(tid<256) for(k=0;k<3;k++) s_fi[tid][k]+=s_fi[tid+256][k];
#endif
#if NTHREOPT>=256 && NTHREOPT2<=128
	__syncthreads();
	if (tid < 128)
		for (k = 0; k < 3; k++)
			s_fi[tid][k] += s_fi[tid + 128][k];
#endif
#if NTHREOPT>=128 && NTHREOPT2<=64
	__syncthreads();
	if (tid < 64)
		for (k = 0; k < 3; k++)
			s_fi[tid][k] += s_fi[tid + 64][k];
#endif
#if NTHREOPT>=64 && NTHREOPT2<=32
	__syncthreads();
	if (tid < 32)
		for (k = 0; k < 3; k++)
			s_fi[tid][k] += s_fi[tid + 32][k];
#endif
#if NTHREOPT2<=16
	if (tid < 16)
		for (k = 0; k < 3; k++)
			s_fi[tid][k] += s_fi[tid + 16][k];
#endif
#if NTHREOPT2<=8
	if(tid<8) for(k=0;k<3;k++) s_fi[tid][k]+=s_fi[tid+8][k];
#endif
#if NTHREOPT2<=4
	if(tid<4) for(k=0;k<3;k++) s_fi[tid][k]+=s_fi[tid+4][k];
#endif
#if NTHREOPT2<=2
	if(tid<2) for(k=0;k<3;k++) s_fi[tid][k]+=s_fi[tid+2][k];
#endif
#if NTHREOPT2<=1
	if(tid<1) for(k=0;k<3;k++) s_fi[tid][k]+=s_fi[tid+1][k];
#endif
	if (jdiv == 0)
		for (k = 0; k < 3; k++)
			fvec[i * 3 + k] = s_fi[tid][k];
#endif
}

__global__
void velforce_kernel(int n3, float *fc, float *a_mass, float *vl,
                     VG_XVEC *atype, int *atype_mat, float hsq,float *ekin1,
                     float *poss, float *sideh){
#ifdef KER
	__shared__ float cache [ThreadsPB];
    int indx = threadIdx.x;
	int tid  = threadIdx.x + blockIdx.x * blockDim.x;

	cache [indx] = 0;

	if (tid < n3 ){
		fc[tid]-= fc[tid]/(n3/3);
		fc[tid] *= hsq/a_mass[atype_mat[atype[tid/3].atype]];
		cache [indx] = vl[tid]*vl[tid]*a_mass[atype_mat[atype[tid/3].atype]];
#ifdef INTEROP
		poss[tid] = atype[tid / 3].r[tid % 3]-sideh[tid % 3]; // for graphics VBO -- Position
#endif
	}
	__syncthreads();

	for (unsigned int s=blockDim.x/2; s>0; s>>=1)
	{
		if (indx < s)
		{
			cache[indx] += cache[indx + s];
		}
		__syncthreads();
	}
	if (indx == 0) ekin1[blockIdx.x] = cache [0];

#endif
}

__global__
void reduction (float *ekin,float *mtemp,float *mpres,float *xs,float tscale,
                    float nden, float vir,int s_num,int w_num,float rtemp,
					float lq,float hsq,float *ekin1, int limi){

#ifdef KER
	__shared__ float cache [NTHREOPT];

  int indx = threadIdx.x;

	cache [indx] = (indx < limi) ? ekin1[indx]:0.0f;

	__syncthreads();

	for (unsigned int s=NTHREOPT/2; s>0; s>>=1){
		if (indx < s)
		{
			cache[indx] += cache[indx + s];
		}
			__syncthreads();
	  }

	if (indx == 0){
		*ekin = cache [0];
		*ekin /= hsq;
		*mtemp = tscale * (*ekin);
		*mpres  = nden / 3.f * ((*ekin) - (vir)) / (s_num + w_num);
		*xs += (*mtemp - rtemp) /  lq * hsq *.5f;
	}

#endif
}

#ifdef INTEROP
__global__
void colorn4(int n4,float *vl,VG_XVEC *atype, int *atype_mat, float *colorvbo){
#ifdef KER
	int tid  = threadIdx.x + blockIdx.x * blockDim.x;
	float d0;
	float d0aux[4];

	d0 = (vl[tid/4]*vl[tid/4]+vl[tid/4+1]*vl[tid/4+1]+vl[tid/4+2]*vl[tid/4+2])*500;
	d0aux[0] 	= d0;
	d0aux[1] 	= d0/3;
	d0aux[2]	= d0/3;
	d0aux[3]	= 0;

	if (tid < n4){
		colorvbo[tid] = d_color_table[atype_mat[atype[tid/4].atype]][tid%4] + d0aux[tid%4];
	}
#endif
}
#endif


#ifdef DP
__global__
void md_loop_cuda (	int n3, float *vl,VG_XVEC *cd,float *xs,float *fc,float *side,
					int n, int nat, float xmax,
					float *a_mass, int *atype_mat, float hsq,float *ekin1,
					float *ekin,float *mtemp,float *mpres,float tscale,
					  float nden, float vir,int s_num,int w_num,float rtemp,
					  float lq,int limi,
					  int md_step, float *poss, float *sideh, float *colorvbo)
{
#if 1
	int  blocksPGrid = (n3 + ThreadsPB - 1)/(ThreadsPB);
	dim3 THREADS(NTHRE);
	dim3 BLOCKS((n3 + ThreadsPB - 1)/(ThreadsPB));
	dim3 threads(NTHREOPT);
	dim3 grid((n * NDIV + NTHREOPT - 1) / NTHREOPT);
	dim3 colorgridn4(((n*4) + ThreadsPB - 1)/(ThreadsPB));

	for(int md_loop = 0; md_loop < md_step; md_loop++){
		update_coor_kernel<<<BLOCKS,THREADS>>>(n3,vl,cd,xs,fc,side);
		nacl_kernel_if2<<<grid, threads>>>(cd, n, nat, xmax, fc);
		velforce_kernel<<<BLOCKS,THREADS>>>(n3,fc,a_mass,vl,cd,atype_mat,hsq,ekin1,poss,sideh);
		reduction<<<1,NTHRE>>>(ekin,mtemp,mpres,xs,tscale,nden,vir,s_num,w_num,rtemp,lq,hsq,ekin1,blocksPGrid);
	}
#ifdef INTEROP
	colorn4<<<colorgridn4,THREADS>>>(n*4,vl,cd,atype_mat,colorvbo);
#endif

#endif
}

#endif
//////////////////NaCl Optmized
///////////////////////////////

extern "C"
void mdlop(int n3,int grape_flg,double phi [3],double *phir,double *iphi, double *vir,int s_num3,
			timeval time_v,double *md_time0,double *md_time,int *m_clock,int md_step,double *mtemp,
			double tscale,double *mpres,double nden,int s_num,int w_num,double rtemp,double lq,
			double x[], int n, int atype[], int nat,
			double pol[], double sigm[], double ipotro[],
		 	double pc[], double pd[],double zz[],
		 	int tblno, double xmax, int periodicflag,
		 	double force[],
			double hsq,double a_mass [], int atype_mat [], double *ekin,double *vl,
			double *xs,double side [],int *firstmalloc, double sideh[]){

//////////////VARIABLES FROM THE BEGINING/////////////////
	//int md_loop;
	//float *forcef=NULL;
	int i,j;
	float xmaxf;
  if((periodicflag & 1)==0) xmax*=2.0;
	xmaxf=xmax;
  int n4 = n*4;
/////////////////////////////////////////////////////////
	int  blocksPGrid = (n3 + ThreadsPB - 1)/(ThreadsPB);
	dim3 THREADS(NTHRE);
	dim3 BLOCKS((n3 + ThreadsPB - 1)/(ThreadsPB));
	dim3 threads(NTHREOPT);
	dim3 grid((n * NDIV + NTHREOPT - 1) / NTHREOPT);
	dim3 colorgridn4((n4 + ThreadsPB - 1)/(ThreadsPB));

	float   fxs = *xs;
	float   fside[3],*ffc, fsideh[3];
	float   *vla;
	VG_XVEC	*veca;

	int     p = 0;
	float   hsqf = hsq;
	float   *fvl,fa_mass[4];

	float ftscale = tscale,fnden = nden,frtemp = rtemp,flq = lq,fvir = 0;
	float fmtemp = *mtemp,fmpres = *mpres;

	vla		= (float*)	malloc(n3*sizeof(float));
	veca  = (VG_XVEC*)malloc((n+NTHREOPT2)*sizeof(VG_XVEC));


	if(*firstmalloc == 0){

		printf("CUDA malloc time...\n");

		// Allocating memory for float conversion.
		ffc = (float*)		malloc(n3*sizeof(float));
		fvl = (float*)		malloc(n3*sizeof(float));
		vec = (VG_XVEC*) 	malloc((NMAX+NTHREOPT2)*sizeof(VG_XVEC));

		// Conversion from Double to Float
		for (p=0;p<4;p++) fa_mass[p] = (float) a_mass[p];
		for (p=0;p<3;p++) fside[p] 	 = (float) side[p];
		for (p=0;p<3;p++) fsideh[p]  = (float) sideh[p];
		for (p=0;p<n3;p++){
			fvl     [p] =  (float) *(vl +p);
			ffc     [p] =  (float) *(force +p);
		}

		for (i = 0; i < (n + NTHREOPT2 - 1) / NTHREOPT2 * NTHREOPT2; i++) {
			if (i < n) {
				for (j = 0; j < 3; j++) {
					vec[i].r[j] = x[i * 3 + j];
				}
				vec[i].atype = atype[i];
			}
			else {
				for (j = 0; j < 3; j++) {
					vec[i].r[j] = 0.0f;
				}
				vec[i].atype = 0;
			}
		}

		// Free CUDA memory. In case we already allocate
		checkCudaErrors(hipFree(d_x));
		checkCudaErrors(hipFree(d_force));
		checkCudaErrors(hipFree(d_side));
		checkCudaErrors(hipFree(d_sideh));
		checkCudaErrors(hipFree(d_amass));
		checkCudaErrors(hipFree(d_vl));
		checkCudaErrors(hipFree(d_atypemat));
		checkCudaErrors(hipFree(d_ekin));
		checkCudaErrors(hipFree(d_xs));
		checkCudaErrors(hipFree(d_mtemp));
		checkCudaErrors(hipFree(d_mpres));
		checkCudaErrors(hipFree(d_ekin1));


		// Allocate global memory to GPU
		checkCudaErrors(hipMalloc((void**)&d_x,sizeof(VG_XVEC)* (NMAX + NTHREOPT2)));
		checkCudaErrors(hipMalloc((void**)&d_force,sizeof(float)*(NMAX + NTHREOPT2)*3));
		checkCudaErrors(hipMalloc((void**)&d_side,3*sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&d_sideh,3*sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&d_amass,4*sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&d_vl,n3*sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&d_atypemat,20*sizeof(int)));
		checkCudaErrors(hipMalloc((void**)&d_ekin,sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&d_xs,sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&d_mtemp,sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&d_mpres,sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&d_ekin1,blocksPGrid*sizeof(float)));
		//checkCudaErrors(hipMalloc((void**)&d_poss,n3*sizeof(float)));

		// Copy memory from CPU to GPU
		checkCudaErrors(hipMemcpy(d_x,vec,sizeof(VG_XVEC)*((n + NTHREOPT2 - 1) / NTHREOPT2 * NTHREOPT2),hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_side,fside,sizeof(float)*3,hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_sideh,fsideh,sizeof(float)*3,hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_mtemp,&fmtemp,sizeof(float),hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_mpres,&fmpres,sizeof(float),hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_xs,&fxs,sizeof(float),hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_vl,fvl,sizeof(float)*n3,hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_amass,fa_mass,sizeof(float)*4,hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_atypemat,atype_mat,sizeof(int)*20,hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_force,ffc,sizeof(float)*n*3,hipMemcpyHostToDevice));

		// Free the memory used to convert to Float
		free(ffc);
		free(fvl);
		free(vec);
	}

#ifdef INTEROP
	//Interoperability
	// Position
	size_t vbosizepos;
	checkCudaErrors(hipGraphicsMapResources(1,&g_strucPossVBOCUDA,0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&d_poss,
																												&vbosizepos,
																												g_strucPossVBOCUDA));
	// Color
	size_t vbosizecol;
	checkCudaErrors(hipGraphicsMapResources(1,&g_strucColorVBOCUDA,0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&d_colr,
																												&vbosizecol,
																												g_strucColorVBOCUDA));
#endif

	///////Md_loop///////////////////////////////////////////////

#ifdef DP
#ifndef TIME_MEMORY
	gettimeofday(&time_v,NULL);
	*md_time0 = (time_v.tv_sec + time_v.tv_usec / 1000000.0);
#endif
//	for (int m=0;m<1000;m++){
	md_loop_cuda<<<1,1>>>(n3,d_vl,d_x,d_xs,d_force,d_side,
						n,nat,xmaxf,
						d_amass,d_atypemat,hsqf,d_ekin1,
						d_ekin,d_mtemp,d_mpres,ftscale,fnden,fvir,s_num,w_num,frtemp,flq,blocksPGrid,
						md_step,d_poss,d_sideh,d_colr);
	//}
	*m_clock+=md_step;
	hipDeviceSynchronize();

#ifndef TIME_MEMORY
	gettimeofday(&time_v,NULL);
	*md_time = (time_v.tv_sec + time_v.tv_usec / 1000000.0);
#endif

#else
	gettimeofday(&time_v,NULL);
	*md_time0 = (time_v.tv_sec + time_v.tv_usec / 1000000.0);
//	for (int m=0;m<10000;m++){
	for(int md_loop = 0; md_loop < md_step; md_loop++){
		update_coor_kernel<<<BLOCKS,THREADS>>>(n3,d_vl,d_x,d_xs,d_force,d_side);
		nacl_kernel_if2<<<grid, threads>>>(d_x, n, nat, xmaxf, d_force);
		velforce_kernel<<<BLOCKS,THREADS>>>(n3,d_force,d_amass,d_vl,d_x,d_atypemat,hsqf,d_ekin1,d_poss,d_sideh);
		reduction<<<1,threads>>>(d_ekin,d_mtemp,d_mpres,d_xs,ftscale,fnden,fvir,s_num,w_num,frtemp,flq,hsqf,d_ekin1,blocksPGrid);
	}
#ifdef INTEROP
	colorn4<<<colorgridn4,THREADS>>>(n4,d_vl,d_x,d_atypemat,d_colr); // Just update after the cycle. For color output.
#endif
//	}
	*m_clock+=md_step;
	hipDeviceSynchronize();
	gettimeofday(&time_v,NULL);
	*md_time = (time_v.tv_sec + time_v.tv_usec / 1000000.0);
#endif

/////////////////Copy back to the CPU
	//CUDA_SAFE_CALL(hipMemcpy(forcef,d_force,sizeof(float)*n*3,hipMemcpyDeviceToHost));
	//CUDA_SAFE_CALL(hipMemcpy(&fxs,d_xs,sizeof(float),hipMemcpyDeviceToHost));
	//CUDA_SAFE_CALL(hipMemcpy(&ekinaux,d_ekin,sizeof(float),hipMemcpyDeviceToHost));
	//CUDA_SAFE_CALL(hipMemcpy(&fmtemp,d_mtemp,sizeof(float),hipMemcpyDeviceToHost));
	//CUDA_SAFE_CALL(hipMemcpy(&fmpres,d_mpres,sizeof(float),hipMemcpyDeviceToHost));

#ifdef TIME_MEMORY
	gettimeofday(&time_v,NULL);
	*md_time0 = (time_v.tv_sec + time_v.tv_usec / 1000000.0);
#endif

#ifdef INTEROP
	checkCudaErrors(hipGraphicsUnmapResources(1,&g_strucPossVBOCUDA,0));
	checkCudaErrors(hipGraphicsUnmapResources(1,&g_strucColorVBOCUDA,0));
#endif

#ifndef INTEROP
	checkCudaErrors(hipMemcpy(vla,d_vl,n3*sizeof(float),hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(veca,d_x,n*sizeof(VG_XVEC),hipMemcpyDeviceToHost));
	//checkCudaErrors(hipMemcpy(cord,d_poss,n3*sizeof(float),hipMemcpyDeviceToHost));
#endif

#ifdef TIME_MEMORY
	gettimeofday(&time_v,NULL);
	*md_time = (time_v.tv_sec + time_v.tv_usec / 1000000.0);
#endif


	//for(i=0;i<n;i++) for(j=0;j<3;j++) force[i*3+j]=(double) forcef[i*3+j];
	for(p=0;p<n3;p++) *(vl+p) = (double) vla[p];
	for(i=0;i<n;i++)for(j=0;j<3;j++) *(x+i*3+j) = (double)veca[i].r[j];
	//for(i=0;i<n;i++)for(j=0;j<3;j++) *(x+i*3+j) = (double)cord[j+i*3];


	free(veca);
	free(vla);
	//free(cord);
	*firstmalloc = 1;

}



